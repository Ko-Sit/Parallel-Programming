#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <cutil_inline.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <fstream>

#define ASIZE 256
#define DATA_SIZE 1024

__device__ int shifts[ASIZE];
__device__ int results[DATA_SIZE];

__global__ void processPattern(char* x ,int m, int shifts[])
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if ( idx >= m ) return;

	char c = x[idx];
	for( int i = m - 1; i >= idx; --i )
	{
		if ( x[i] == c )
		{// match is found
			shifts[c] = m - i;
			return;
		}
	}
}

__global__ void search(char *x, int m, char* y, int n, int shifts[], int indx[], int results[])
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx > (n - m) )
		return;
	if ( indx[idx] != idx )
		return;

	unsigned int yes = 1;
	for( int i = 0; i < m; ++i )
	{
		// try to match the string
		if ( x[i] != y[idx + i] )
		{
			yes = 0;
			break;
		}
	}
	results[idx] = yes;
}

void precomputeShiftIndx(char* y, int n, int m, int shifts[], int indx[])
{
	int j = 0;
	int limit = n - m;

	while (j <= limit ) {
		j += shifts[ y[j + m] ];
		indx[j] = j;
	}
}


void display_results(int n, int  results[]) {
	int j = 0;
	int flag =0;
	for( int i =0; i < n; ++i )
		if ( results[i] == 1 )
		{
			printf("%d. Found match at %d\n",j++, i);
			flag=1;
		}
	if(flag==0)
		printf("Not found\n");
}

int main(int argc, char* argv[])
{
	srand(time(NULL));
	char values[] = "ACGT";
	int cuda_device = 0;
	int n = 10000000; // length of main string
	int m = 100; // length of substring


	char* mainString = (char*)malloc(n * sizeof(char));
	char* subString = (char*)malloc(m * sizeof(char));
	for(int i=0;i < n;i++)
	{
		mainString[i] = values[rand()%4];
	}
	for(int i=0;i < m;i++)
	{
		subString[i] = values[rand()%4];
	}

	//
	// Initialize the shift and index array
	//
	int* l_shifts = (int*)malloc( ASIZE * sizeof(int) );
	for( int i = 0; i < ASIZE; ++i )
		l_shifts[i] = m + 1;
	int* l_indx = (int*) malloc( n * sizeof(int) );
	for( int i = 0; i < n; ++i )
		l_indx[i] = -1;

	hipError_t error;
	hipEvent_t start_event, stop_event;
	float time;
	float time2;

	// initializing the GPU timers
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);


	//
	// Allocate global memory to host the pattern, text and other supporting data
	// structures
	//
	char* d_substr = 0;
	int* d_shifts = 0;
	int* d_indx = 0;
	char* d_text = 0;
	int *d_results = 0,*l_results=(int*) malloc( n * sizeof(int) );
	for( int i = 0; i < n; ++i )
		l_results[i] = 0;
	//hipGetSymbolAddress((void**)&d_shifts, "shifts");
	hipMalloc((void**)&d_results, n * sizeof(int)) ;

	hipMalloc((void**)&d_shifts, sizeof(int) * ASIZE) ;
	//error = hipGetLastError();
	//printf("Error1: %s\n", hipGetErrorString(error));
	hipMalloc((void**)&d_indx, n * sizeof(int)) ;
	//error = hipGetLastError();
	//printf("Error2: %s\n", hipGetErrorString(error));
	hipMalloc((void**)&d_substr, (m + 1)*sizeof(char)) ;
	//error = hipGetLastError();
	//printf("Error3: %s\n", hipGetErrorString(error));
	hipMalloc((void**)&d_text, (strlen(mainString)+1)*sizeof(char)) ;
	//error = hipGetLastError();
	//printf("Error4: %s\n", hipGetErrorString(error));
	hipMemcpy(d_shifts, l_shifts, sizeof(int) * ASIZE, hipMemcpyHostToDevice ) ;
	hipMemcpy(d_results, l_results, sizeof(int) * n, hipMemcpyHostToDevice ) ;
	//error = hipGetLastError();
	//printf("Error5: %s\n", hipGetErrorString(error));
	hipMemcpy(d_text, mainString, sizeof(char)*(strlen(mainString)+1), hipMemcpyHostToDevice ) ;
	//error = hipGetLastError();
	//printf("Error6: %s\n", hipGetErrorString(error));
	hipMemcpy(d_substr, subString, sizeof(char)*(strlen(subString)+1), hipMemcpyHostToDevice) ;
	//error = hipGetLastError();
	//printf("Error7: %s\n", hipGetErrorString(error));

	//
	// Pre-process the pattern to be matched
	//
	dim3 threadsPerBlocks(ASIZE, 1);
	int t = m / threadsPerBlocks.x;
	int t1 = m % threadsPerBlocks.x;
	if ( t1 != 0 ) t += 1;
	dim3 numBlocks(t, 1);

	printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks.x, threadsPerBlocks.x);
	hipEventRecord(start_event, 0);
	processPattern<<<numBlocks,threadsPerBlocks>>>(d_substr, m, d_shifts);
	hipDeviceSynchronize();

	hipEventRecord(stop_event, 0);
	hipEventSynchronize( stop_event );
	hipEventElapsedTime( &time, start_event, stop_event );

	hipMemcpy(l_shifts, d_shifts, sizeof(int) * ASIZE, hipMemcpyDeviceToHost ) ;
	//error = hipGetLastError();
	//printf("Error8: %s\n", hipGetErrorString(error));
	//
	// Transfer the pre-computed shift indexes from host to device memory
	//
	hipMemcpy(l_shifts, d_shifts, ASIZE * sizeof(int), hipMemcpyDeviceToHost) ;
	precomputeShiftIndx(mainString , n, m, l_shifts, l_indx);
	hipMemcpy(d_shifts, l_shifts, ASIZE * sizeof(int), hipMemcpyHostToDevice) ;
	hipMemcpy(d_indx, l_indx, n * sizeof(int), hipMemcpyHostToDevice) ;
	//error = hipGetLastError();
	//printf("Error9: %s\n", hipGetErrorString(error));
	//
	// Perform the actual search
	//
	t = n / threadsPerBlocks.x;
	t1 = n % threadsPerBlocks.x;
	if ( t1 != 0 ) t += 1;
	dim3 numBlocks2(t, 1);
	printf("Launching kernel with blocks=%d, threadsperblock=%d\n", numBlocks2.x, threadsPerBlocks.x);
	hipEventRecord(start_event, 0);
	search<<<numBlocks2,threadsPerBlocks>>>(d_substr, m, d_text, n, d_shifts, d_indx,d_results);
	hipDeviceSynchronize();

	hipEventRecord(stop_event, 0);
	hipEventSynchronize( stop_event );
	hipEventElapsedTime( &time2, start_event, stop_event );

	hipEventDestroy( start_event ); // cleanup
	hipEventDestroy( stop_event ); // cleanup
	printf("done and it took: %f+%f=%f milliseconds\n",time, time2, time+time2);

	//hipGetSymbolAddress((void**)&d_results, "results");
	//hipMalloc((void**)&d_results, n * sizeof(int)) ;
	//int* l_results = (int*) malloc( n * sizeof(int) );
	hipMemcpy(l_results, d_results, n * sizeof(int), hipMemcpyDeviceToHost) ;
	display_results(n, l_results);
	//error = hipGetLastError();
	//printf("Error10: %s\n", hipGetErrorString(error));
	hipFree(d_substr);
	hipFree(d_shifts);
	hipFree(d_indx);
	hipFree(d_text);
	free(mainString);
	free(subString);
	free(l_indx);
	free(l_shifts);
	free(l_results);

	hipDeviceReset();

	return 0;
}
